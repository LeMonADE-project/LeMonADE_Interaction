#include "hip/hip_runtime.h"
/*--------------------------------------------------------------------------------
    ooo      L   attice-based  |
  o\.|./o    e   xtensible     | LeMonADE: An Open Source Implementation of the
 o\.\|/./o   Mon te-Carlo      |           Bond-Fluctuation-Model for Polymers
oo--GPU--oo  A   lgorithm and  |
 o/./|\.\o   D   evelopment    | Copyright (C) 2013-2015 by
  o/.|.\o    E   nvironment    | LeMonADE Principal Developers (see AUTHORS)
    ooo                        |
----------------------------------------------------------------------------------

This file is part of LeMonADEGPU.

LeMonADE is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

LeMonADE is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with LeMonADE.  If not, see <http://www.gnu.org/licenses/>.

--------------------------------------------------------------------------------*/
/*
 * UpdaterGPU_Interaction.cu
 *
 *  Created on: 04.02.2022
 *     Authors: Toni Mueller
 */
#include <LeMonADE_Interaction/updater/UpdaterGPU_Interaction.h>

#include <LeMonADEGPU/utility/cudacommon.hpp>
#include <LeMonADEGPU/core/Method.h>
#include <LeMonADEGPU/utility/DeleteMirroredObject.h>
#include <hip/hip_runtime_api.h>              // hipProfilerStop
#include <LeMonADEGPU/utility/AutomaticThreadChooser.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>
#include <extern/Fundamental/BitsCompileTime.hpp>
#include <LeMonADEGPU/utility/cudacommon.hpp>
#include <LeMonADEGPU/utility/SelectiveLogger.hpp>
#include <LeMonADEGPU/utility/graphColoring.tpp>
#include <LeMonADEGPU/utility/graphColoring.h>
#include <LeMonADEGPU/core/rngs/Saru.h>
#include <LeMonADEGPU/core/MonomerEdges.h>
#include <LeMonADEGPU/core/constants.cuh>
#include <LeMonADEGPU/feature/BoxCheck.h>
#include <LeMonADEGPU/core/Method.h>
#include <LeMonADEGPU/utility/DeleteMirroredObject.h>
#include <LeMonADEGPU/core/BondVectorSet.h>
using T_Flags            = UpdaterGPU_Interaction< uint8_t >::T_Flags         ;
using T_Id               = UpdaterGPU_Interaction< uint8_t >::T_Id            ;
using T_InteractionTag   = UpdaterGPU_Interaction< uint8_t >::T_InteractionTag;
using T_Color            = UpdaterGPU_Interaction< uint8_t >::T_Color         ;
__device__ __constant__ uint32_t DXTableNN_d[18];
__device__ __constant__ uint32_t DYTableNN_d[18];
__device__ __constant__ uint32_t DZTableNN_d[18];
__device__ __constant__ double dcNNProbability[32][32];
__global__ void  kernelPrintTagType(){

    // auto T_Id id = blockIdx.x * blockDim.x + threadIdx.x;
    printf("TagType[%d][%d]=%f\n",blockIdx.x,  threadIdx.x, dcNNProbability[blockIdx.x][threadIdx.x] );
}
/**
 * @brief convinience function to print the box dimensions for the device constants 
 */
 __global__ void CheckBoxDimensions()
 {
 printf("KernelCheckBoxDimensions: %d %d %d %d %d %d  %d %d \n",dcBoxX, dcBoxY, dcBoxZ,dcBoxXM1, dcBoxYM1,dcBoxZM1, dcBoxXLog2, dcBoxXYLog2 );
 }
 __global__ void checkCurve(
     Method const met
 ){
    uint32_t id = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t x(met.getCurve().linearizeBoxVectorIndexX(id));
    uint32_t y(met.getCurve().linearizeBoxVectorIndexY(id));
    uint32_t z(met.getCurve().linearizeBoxVectorIndexZ(id));

    uint32_t xM2(met.getCurve().linearizeBoxVectorIndexX(id+(0u-2u)));
    uint32_t yM2(met.getCurve().linearizeBoxVectorIndexY(id+(0u-2u)));
    uint32_t zM2(met.getCurve().linearizeBoxVectorIndexZ(id+(0u-2u)));

    uint32_t xP2(met.getCurve().linearizeBoxVectorIndexX(id+2u));
    uint32_t yP2(met.getCurve().linearizeBoxVectorIndexY(id+2u));
    uint32_t zP2(met.getCurve().linearizeBoxVectorIndexZ(id+2u));

    printf("%d (%d %d %d) (%d %d %d) (%d %d %d) %d \n", id, x,y,z,xM2,yM2,zM2,xP2,yP2,zP2,( (-2) & dcBoxXM1 )  );
}
///////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////
///////Defintion of member functions for the interaction lattice //////////////
///////////////////////////////////////////////////////////////////////////////
template< typename T_UCoordinateCuda >
void UpdaterGPU_Interaction<T_UCoordinateCuda>::initializeInteractionLattice(){
    if ( mLatticeInteractionTag != NULL ){
        std::stringstream msg;
        msg << "[" << __FILENAME__ << "::initializeInteractionLattice] "
            << "Initialize was already called and may not be called again "
            << "until cleanup was called!";
        mLog( "Error" ) << msg.str();
        throw std::runtime_error( msg.str() );
    }
    size_t nBytesLatticeTmp = mBoxX * mBoxY * mBoxZ * sizeof(T_InteractionTag);
     mLog( "Info" ) << "Allocate "<< nBytesLatticeTmp/1024<<"kB  memory for lattice \n";  
    mLatticeInteractionTag  = new MirroredTexture< T_InteractionTag >( nBytesLatticeTmp, mStream );
}

/**
 * @brief writes the ID of the chain ends on the lattice
 * @details The ID start at 1 and are shifted by and offset which is given
 * 	    by the previous species of monomers. 
 */
template< typename T_UCoordinateCuda >
__global__ void kernelUpdateInteractionLattice
(
    typename CudaVec4< T_UCoordinateCuda >::value_type
                        const * const __restrict__ dpPolymerSystem     ,
    T_InteractionTag    const * const __restrict__ dInteractionTag     ,                
    uint32_t            const                      iOffset             ,
    T_InteractionTag          * const __restrict__ dpInteractionLattice,
    T_Id                        const              nMonomers           ,
    Method                      const              met 
){
    for ( T_Id id = blockIdx.x * blockDim.x + threadIdx.x;
          id < nMonomers; id += gridDim.x * blockDim.x ){
        
        auto const r0 = dpPolymerSystem[ iOffset + id ];
        T_InteractionTag const interactionTag( dInteractionTag[iOffset + id] + T_InteractionTag(1) );
        uint32_t x=r0.x;
        uint32_t y=r0.y;
        uint32_t z=r0.z;

        auto const x0Abs  = met.getCurve().linearizeBoxVectorIndexX( x               );
        auto const x0POne = met.getCurve().linearizeBoxVectorIndexX( x + uint32_t(1) );
    
        auto const y0Abs  = met.getCurve().linearizeBoxVectorIndexY( y                );
        auto const y0POne = met.getCurve().linearizeBoxVectorIndexY( y  + uint32_t(1) );
    
        auto const z0Abs  = met.getCurve().linearizeBoxVectorIndexZ( z                );
        auto const z0POne = met.getCurve().linearizeBoxVectorIndexZ( z  + uint32_t(1) );
        
        if (
            dpInteractionLattice[ x0Abs  + y0Abs  + z0Abs  ] != T_InteractionTag(0) ||
            dpInteractionLattice[ x0Abs  + y0POne + z0Abs  ] != T_InteractionTag(0) ||
            dpInteractionLattice[ x0Abs  + y0Abs  + z0POne ] != T_InteractionTag(0) ||
            dpInteractionLattice[ x0Abs  + y0POne + z0POne ] != T_InteractionTag(0) ||
            dpInteractionLattice[ x0POne + y0Abs  + z0Abs  ] != T_InteractionTag(0) ||
            dpInteractionLattice[ x0POne + y0POne + z0Abs  ] != T_InteractionTag(0) ||
            dpInteractionLattice[ x0POne + y0Abs  + z0POne ] != T_InteractionTag(0) ||
            dpInteractionLattice[ x0POne + y0POne + z0POne ] != T_InteractionTag(0) 
        ) {
            printf("Occupy an already occupied lattice site: %d %d %d %d %d %d %d %d\n",   
            dpInteractionLattice[ x0Abs  + y0Abs  + z0Abs  ] ,
            dpInteractionLattice[ x0Abs  + y0POne + z0Abs  ] ,
            dpInteractionLattice[ x0Abs  + y0Abs  + z0POne ] ,
            dpInteractionLattice[ x0Abs  + y0POne + z0POne ] ,
            dpInteractionLattice[ x0POne + y0Abs  + z0Abs  ] ,
            dpInteractionLattice[ x0POne + y0POne + z0Abs  ] ,
            dpInteractionLattice[ x0POne + y0Abs  + z0POne ] ,
            dpInteractionLattice[ x0POne + y0POne + z0POne ] );
        }
        dpInteractionLattice[ x0Abs  + y0Abs  + z0Abs  ] = interactionTag;
        dpInteractionLattice[ x0Abs  + y0POne + z0Abs  ] = interactionTag;
        dpInteractionLattice[ x0Abs  + y0Abs  + z0POne ] = interactionTag;
        dpInteractionLattice[ x0Abs  + y0POne + z0POne ] = interactionTag;
        dpInteractionLattice[ x0POne + y0Abs  + z0Abs  ] = interactionTag;
        dpInteractionLattice[ x0POne + y0POne + z0Abs  ] = interactionTag;
        dpInteractionLattice[ x0POne + y0Abs  + z0POne ] = interactionTag;
        dpInteractionLattice[ x0POne + y0POne + z0POne ] = interactionTag;
    }
}
 /**
  * @brief convinience function to update the lattice occupation. 
  * @details We introduce such functions because then they can be used latter on from inheritate classes..
  */
template< typename T_UCoordinateCuda >
void UpdaterGPU_Interaction< T_UCoordinateCuda >::launch_initializeInteractionLattice(
  const size_t nBlocks , const size_t nThreads, const T_Id iSpecies ){
	// mLog ( "Check" ) <<"Start filling lattice with ones:  \n" ;
	if ( false ){ //fill in cpu 
		mPolymerSystemSorted->pop();
		for (T_Id i =0; i < mnElementsInGroup[ iSpecies ]; i++){
			auto const id(i+mviSubGroupOffsets[ iSpecies ]);
			auto const r(mPolymerSystemSorted->host[id]); 
			auto const Vector(met.getCurve().linearizeBoxVectorIndex(r.x,r.y,r.z));
			mLatticeInteractionTag->host[Vector]= mInteractionTag->host[id]+1;
		}
		mLatticeInteractionTag->push(0);
		hipStreamSynchronize( mStream );
	}else{
		kernelUpdateInteractionLattice<T_UCoordinateCuda><<<nBlocks,nThreads,0,mStream>>>(
			mPolymerSystemSorted->gpu     ,         
            mInteractionTag->gpu          ,   
			mviSubGroupOffsets[ iSpecies ], 
			mLatticeInteractionTag->gpu   ,
			mnElementsInGroup[ iSpecies ] ,                        
			met
		);
	}
}
/**
 * @brief writes 0 on the lattice where the chain ends are located 
 * @details Using this brings performance, because the lattice is dilute
 */
template< typename T_UCoordinateCuda >
__global__ void kernelResetInteractionLattice
(
    typename CudaVec4< T_UCoordinateCuda >::value_type
                        const * const __restrict__ dpPolymerSystem     ,
    T_InteractionTag    const * const __restrict__ dInteractionTag     ,
    uint32_t            const                      iOffset             ,
    T_InteractionTag          * const __restrict__ dpInteractionLattice,
    T_Id                        const              nMonomers           ,
    Method                      const              met 
){
    for ( T_Id id = blockIdx.x * blockDim.x + threadIdx.x;
        id < nMonomers; id += gridDim.x * blockDim.x ){
        
        auto const r0 = dpPolymerSystem[ iOffset + id ];
        T_InteractionTag const interactionTagReset(0);
        T_InteractionTag const interactionTag( dInteractionTag[iOffset + id] + T_InteractionTag(1) );
        uint32_t x=r0.x;
        uint32_t y=r0.y;
        uint32_t z=r0.z;

        auto const x0Abs  = met.getCurve().linearizeBoxVectorIndexX( x               );
        auto const x0POne = met.getCurve().linearizeBoxVectorIndexX( x + uint32_t(1) );
    
        auto const y0Abs  = met.getCurve().linearizeBoxVectorIndexY( y                );
        auto const y0POne = met.getCurve().linearizeBoxVectorIndexY( y  + uint32_t(1) );
    
        auto const z0Abs  = met.getCurve().linearizeBoxVectorIndexZ( z                );
        auto const z0POne = met.getCurve().linearizeBoxVectorIndexZ( z  + uint32_t(1) );
        if (
            dpInteractionLattice[ x0Abs  + y0Abs  + z0Abs  ] != interactionTag ||
            dpInteractionLattice[ x0Abs  + y0POne + z0Abs  ] != interactionTag ||
            dpInteractionLattice[ x0Abs  + y0Abs  + z0POne ] != interactionTag ||
            dpInteractionLattice[ x0Abs  + y0POne + z0POne ] != interactionTag ||
            dpInteractionLattice[ x0POne + y0Abs  + z0Abs  ] != interactionTag ||
            dpInteractionLattice[ x0POne + y0POne + z0Abs  ] != interactionTag ||
            dpInteractionLattice[ x0POne + y0Abs  + z0POne ] != interactionTag ||
            dpInteractionLattice[ x0POne + y0POne + z0POne ] != interactionTag 
        ) {
            printf("Occupy an already occupied lattice site: %d %d %d %d %d %d %d %d\n",   
            dpInteractionLattice[ x0Abs  + y0Abs  + z0Abs  ] ,
            dpInteractionLattice[ x0Abs  + y0POne + z0Abs  ] ,
            dpInteractionLattice[ x0Abs  + y0Abs  + z0POne ] ,
            dpInteractionLattice[ x0Abs  + y0POne + z0POne ] ,
            dpInteractionLattice[ x0POne + y0Abs  + z0Abs  ] ,
            dpInteractionLattice[ x0POne + y0POne + z0Abs  ] ,
            dpInteractionLattice[ x0POne + y0Abs  + z0POne ] ,
            dpInteractionLattice[ x0POne + y0POne + z0POne ] );
        }
        dpInteractionLattice[ x0Abs  + y0Abs  + z0Abs  ] = interactionTagReset;
        dpInteractionLattice[ x0Abs  + y0POne + z0Abs  ] = interactionTagReset;
        dpInteractionLattice[ x0Abs  + y0Abs  + z0POne ] = interactionTagReset;
        dpInteractionLattice[ x0Abs  + y0POne + z0POne ] = interactionTagReset;
        dpInteractionLattice[ x0POne + y0Abs  + z0Abs  ] = interactionTagReset;
        dpInteractionLattice[ x0POne + y0POne + z0Abs  ] = interactionTagReset;
        dpInteractionLattice[ x0POne + y0Abs  + z0POne ] = interactionTagReset;
        dpInteractionLattice[ x0POne + y0POne + z0POne ] = interactionTagReset;
    }
}
template< typename T_UCoordinateCuda >
void UpdaterGPU_Interaction< T_UCoordinateCuda >::launch_resetInteractionLattice(
  const size_t nBlocks , const size_t nThreads, const T_Id iSpecies ){
	if ( false ){ //erasse in cpu 
		mPolymerSystemSorted->pop();
		for (T_Id i =0; i < mnElementsInGroup[ iSpecies ]; i++){
			auto const idB(i+mviSubGroupOffsets[ iSpecies ]);
			auto const r(mPolymerSystemSorted->host[idB]); 
			auto const Vector(met.getCurve().linearizeBoxVectorIndex(r.x,r.y,r.z));
			mLatticeInteractionTag->host[Vector]= 0;
		}
		mLatticeInteractionTag->push(0);
		hipStreamSynchronize( mStream );
	}else{
		kernelResetInteractionLattice<T_UCoordinateCuda><<<nBlocks,nThreads,0,mStream>>>(
		mPolymerSystemSorted->gpu     ,  
        mInteractionTag->gpu          ,             
		mviSubGroupOffsets[ iSpecies ], 
		mLatticeInteractionTag->gpu   ,
		mnElementsInGroup[ iSpecies ] ,                        
		met
		);
	}
}
/**
 * @brief Counts the number of occupied lattice sites.
 */
template< typename T_UCoordinateCuda  >
void UpdaterGPU_Interaction< T_UCoordinateCuda >::checkInteractionLatticeOccupation()  
{
	mLatticeInteractionTag->pop(0);
    mPolymerSystemSorted->pop(0);
    miToiNew->pop(0);
    CUDA_ERROR( hipStreamSynchronize( mStream ) );
	uint32_t countLatticeEntries(0);
    uint32_t countLatticeEntriesAType(0);
    uint32_t countLatticeEntriesBType(0);
	for(T_Id x=0; x< mBoxX; x++ )
		for(T_Id y=0; y< mBoxY; y++ )
			for(T_Id z=0; z< mBoxX; z++ ){
                uint32_t tag(mLatticeInteractionTag->host[met.getCurve().linearizeBoxVectorIndex(x,y,z)]);
				if( tag > 0 ){
					countLatticeEntries++;
                    if (tag == 3 )countLatticeEntriesAType++;
                    if (tag == 4 )countLatticeEntriesBType++;

                }
            }
	assert(mnAllMonomers*8 == countLatticeEntries );  
    assert(mnAllMonomers*4 == countLatticeEntriesAType );  
    assert(mnAllMonomers*4 == countLatticeEntriesBType );  
	// mLog( "Check" )
    std::cout 
		<< "checkInteractionLatticeOccupation: \n"
        << "mnAllMonomers*8          = " <<     mnAllMonomers*8 << "\n"
        << "mnAllMonomers*4          = " <<     mnAllMonomers*4 << "\n"
		<< "mnAllMonomers            = " <<       mnAllMonomers << "\n"
		<< "countLatticeEntries      = " << countLatticeEntries << "\n"
        << "countLatticeEntriesAType = " << countLatticeEntriesAType << "\n"
        << "countLatticeEntriesBType = " << countLatticeEntriesBType << std::endl;
    //TODO:teseting the consistency of the lattice and the specied position
	
    for (auto i=0; i < mnAllMonomers; i++){
        auto const r0(mPolymerSystemSorted->host[miToiNew->host[i]]);
        uint32_t x=r0.x;
        uint32_t y=r0.y;
        uint32_t z=r0.z;
        if (
            !(
            (getAttributeTag(i)+1)== static_cast<uint32_t>(mLatticeInteractionTag->host[met.getCurve().linearizeBoxVectorIndex(x  ,y  ,z  )]) &&
            (getAttributeTag(i)+1)== static_cast<uint32_t>(mLatticeInteractionTag->host[met.getCurve().linearizeBoxVectorIndex(x  ,y+1,z  )]) &&
            (getAttributeTag(i)+1)== static_cast<uint32_t>(mLatticeInteractionTag->host[met.getCurve().linearizeBoxVectorIndex(x  ,y  ,z+1)]) &&
            (getAttributeTag(i)+1)== static_cast<uint32_t>(mLatticeInteractionTag->host[met.getCurve().linearizeBoxVectorIndex(x  ,y+1,z+1)]) &&
            (getAttributeTag(i)+1)== static_cast<uint32_t>(mLatticeInteractionTag->host[met.getCurve().linearizeBoxVectorIndex(x+1,y  ,z  )]) &&
            (getAttributeTag(i)+1)== static_cast<uint32_t>(mLatticeInteractionTag->host[met.getCurve().linearizeBoxVectorIndex(x+1,y+1,z  )]) &&
            (getAttributeTag(i)+1)== static_cast<uint32_t>(mLatticeInteractionTag->host[met.getCurve().linearizeBoxVectorIndex(x+1,y  ,z+1)]) &&
            (getAttributeTag(i)+1)== static_cast<uint32_t>(mLatticeInteractionTag->host[met.getCurve().linearizeBoxVectorIndex(x+1,y+1,z+1)]) 
            )
        ){
            std::stringstream error_message;
            error_message << "AttributeTag["<<i<<"]="<<getAttributeTag(i)+1<<"\n";
            error_message << "LatticeEntry["<<x  <<","<<y  <<","<<z  <<"]="<< static_cast<uint32_t>(mLatticeInteractionTag->host[met.getCurve().linearizeBoxVectorIndex(x  ,y  ,z  )])<< "\n"; 
            error_message << "LatticeEntry["<<x  <<","<<y+1<<","<<z  <<"]="<< static_cast<uint32_t>(mLatticeInteractionTag->host[met.getCurve().linearizeBoxVectorIndex(x  ,y+1,z  )])<< "\n"; 
            error_message << "LatticeEntry["<<x  <<","<<y  <<","<<z+1<<"]="<< static_cast<uint32_t>(mLatticeInteractionTag->host[met.getCurve().linearizeBoxVectorIndex(x  ,y  ,z+1)])<< "\n"; 
            error_message << "LatticeEntry["<<x  <<","<<y+1<<","<<z+1<<"]="<< static_cast<uint32_t>(mLatticeInteractionTag->host[met.getCurve().linearizeBoxVectorIndex(x  ,y+1,z+1)])<< "\n"; 
            error_message << "LatticeEntry["<<x+1<<","<<y  <<","<<z  <<"]="<< static_cast<uint32_t>(mLatticeInteractionTag->host[met.getCurve().linearizeBoxVectorIndex(x+1,y  ,z  )])<< "\n"; 
            error_message << "LatticeEntry["<<x+1<<","<<y+1<<","<<z  <<"]="<< static_cast<uint32_t>(mLatticeInteractionTag->host[met.getCurve().linearizeBoxVectorIndex(x+1,y+1,z  )])<< "\n"; 
            error_message << "LatticeEntry["<<x+1<<","<<y  <<","<<z+1<<"]="<< static_cast<uint32_t>(mLatticeInteractionTag->host[met.getCurve().linearizeBoxVectorIndex(x+1,y  ,z+1)])<< "\n"; 
            error_message << "LatticeEntry["<<x+1<<","<<y+1<<","<<z+1<<"]="<< static_cast<uint32_t>(mLatticeInteractionTag->host[met.getCurve().linearizeBoxVectorIndex(x+1,y+1,z+1)])<< "\n"; 
            throw std::runtime_error(error_message.str());
        }
    }
}
///////////////////////////////////////////////////////////////////////////////
//Lattice handling is done ////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////
//
///////////////////////////////////////////////////////////////////////////////
__device__ inline double getProbability(uint32_t typeA, uint32_t typeB){
    return dcNNProbability[typeA][typeB];
}
__device__ inline double calcInteractionProbability(
    T_InteractionTag  * const __restrict__ dpInteractionLattice,
    uint32_t            const & x0        ,
    uint32_t            const & y0        ,
    uint32_t            const & z0        ,
    T_Flags             const & axis      ,
    Method		        const & met       
){
    auto const dx = DXTableNN_d[ axis ];   // 0 or 1 for  -1,1 
    auto const dy = DYTableNN_d[ axis ];   // 0 or 1 for  -1,1 
    auto const dz = DZTableNN_d[ axis ];   // 0 or 1 for  -1,1 

    auto const x0MTwo = met.getCurve().linearizeBoxVectorIndexX( x0 + dx - uint32_t(2) );
    auto const x0MOne = met.getCurve().linearizeBoxVectorIndexX( x0 + dx - uint32_t(1) );
    auto const x0Abs  = met.getCurve().linearizeBoxVectorIndexX( x0 + dx               );
    auto const x0POne = met.getCurve().linearizeBoxVectorIndexX( x0 + dx + uint32_t(1) );
    auto const x0PTwo = met.getCurve().linearizeBoxVectorIndexX( x0 + dx + uint32_t(2) );

    auto const y0MTwo = met.getCurve().linearizeBoxVectorIndexY( y0 + dy - uint32_t(2) );
    auto const y0MOne = met.getCurve().linearizeBoxVectorIndexY( y0 + dy - uint32_t(1) );
    auto const y0Abs  = met.getCurve().linearizeBoxVectorIndexY( y0 + dy               );
    auto const y0POne = met.getCurve().linearizeBoxVectorIndexY( y0 + dy + uint32_t(1) );
    auto const y0PTwo = met.getCurve().linearizeBoxVectorIndexY( y0 + dy + uint32_t(2) );

    auto const z0MTwo = met.getCurve().linearizeBoxVectorIndexZ( z0 + dz - uint32_t(2) );
    auto const z0MOne = met.getCurve().linearizeBoxVectorIndexZ( z0 + dz - uint32_t(1) );
    auto const z0Abs  = met.getCurve().linearizeBoxVectorIndexZ( z0 + dz               );
    auto const z0POne = met.getCurve().linearizeBoxVectorIndexZ( z0 + dz + uint32_t(1) );
    auto const z0PTwo = met.getCurve().linearizeBoxVectorIndexZ( z0 + dz + uint32_t(2) );

    auto typeA(dpInteractionLattice[ x0Abs + y0Abs + z0Abs ] );
    // printf("tagType: %d %d %.10f\n",typeA, dpInteractionLattice[ x0MTwo + y0Abs  + z0Abs  ], getProbability(typeA, dpInteractionLattice[ x0MTwo + y0Abs  + z0Abs  ]));
    double prop(1);
    switch ( axis >> 1 ){
        case 0 : //+-x
            prop*=getProbability(typeA, dpInteractionLattice[ x0MTwo + y0Abs  + z0Abs  ]);
            prop*=getProbability(typeA, dpInteractionLattice[ x0MTwo + y0POne + z0Abs  ]);
            prop*=getProbability(typeA, dpInteractionLattice[ x0MTwo + y0Abs  + z0POne ]);
            prop*=getProbability(typeA, dpInteractionLattice[ x0MTwo + y0POne + z0POne ]);

            prop*=getProbability(typeA, dpInteractionLattice[ x0MOne + y0MOne + z0Abs  ]);
            prop*=getProbability(typeA, dpInteractionLattice[ x0MOne + y0MOne + z0POne ]);
            prop*=getProbability(typeA, dpInteractionLattice[ x0MOne + y0Abs  + z0MOne ]);
            prop*=getProbability(typeA, dpInteractionLattice[ x0MOne + y0Abs  + z0PTwo ]);
            prop*=getProbability(typeA, dpInteractionLattice[ x0MOne + y0POne + z0MOne ]);
            prop*=getProbability(typeA, dpInteractionLattice[ x0MOne + y0POne + z0PTwo ]);
            prop*=getProbability(typeA, dpInteractionLattice[ x0MOne + y0PTwo + z0Abs  ]);
            prop*=getProbability(typeA, dpInteractionLattice[ x0MOne + y0PTwo + z0POne ]);

            prop/=getProbability(typeA, dpInteractionLattice[ x0POne + y0MOne + z0Abs  ]);
            prop/=getProbability(typeA, dpInteractionLattice[ x0POne + y0MOne + z0POne ]);
            prop/=getProbability(typeA, dpInteractionLattice[ x0POne + y0Abs  + z0MOne ]);
            prop/=getProbability(typeA, dpInteractionLattice[ x0POne + y0Abs  + z0PTwo ]);
            prop/=getProbability(typeA, dpInteractionLattice[ x0POne + y0POne + z0MOne ]);
            prop/=getProbability(typeA, dpInteractionLattice[ x0POne + y0POne + z0PTwo ]);
            prop/=getProbability(typeA, dpInteractionLattice[ x0POne + y0PTwo + z0Abs  ]);
            prop/=getProbability(typeA, dpInteractionLattice[ x0POne + y0PTwo + z0POne ]);

            prop/=getProbability(typeA, dpInteractionLattice[ x0PTwo + y0Abs  + z0Abs  ]);
            prop/=getProbability(typeA, dpInteractionLattice[ x0PTwo + y0POne + z0Abs  ]);
            prop/=getProbability(typeA, dpInteractionLattice[ x0PTwo + y0Abs  + z0POne ]);
            prop/=getProbability(typeA, dpInteractionLattice[ x0PTwo + y0POne + z0POne ]);
        
            if(dx==0){prop=1./prop;}
            break;
        case 1 : //+-y
            prop*=getProbability(typeA, dpInteractionLattice[ x0Abs  + y0MTwo + z0Abs  ]);
            prop*=getProbability(typeA, dpInteractionLattice[ x0POne + y0MTwo + z0Abs  ]);
            prop*=getProbability(typeA, dpInteractionLattice[ x0Abs  + y0MTwo + z0POne ]);
            prop*=getProbability(typeA, dpInteractionLattice[ x0POne + y0MTwo + z0POne ]);

            prop*=getProbability(typeA, dpInteractionLattice[ x0MOne + y0MOne + z0Abs  ]);
            prop*=getProbability(typeA, dpInteractionLattice[ x0MOne + y0MOne + z0POne ]);
            prop*=getProbability(typeA, dpInteractionLattice[ x0Abs  + y0MOne + z0MOne ]);
            prop*=getProbability(typeA, dpInteractionLattice[ x0Abs  + y0MOne + z0PTwo ]);
            prop*=getProbability(typeA, dpInteractionLattice[ x0POne + y0MOne + z0MOne ]);
            prop*=getProbability(typeA, dpInteractionLattice[ x0POne + y0MOne + z0PTwo ]);
            prop*=getProbability(typeA, dpInteractionLattice[ x0PTwo + y0MOne + z0Abs  ]);
            prop*=getProbability(typeA, dpInteractionLattice[ x0PTwo + y0MOne + z0POne ]);

            prop/=getProbability(typeA, dpInteractionLattice[ x0MOne + y0POne + z0Abs  ]);
            prop/=getProbability(typeA, dpInteractionLattice[ x0MOne + y0POne + z0POne ]);
            prop/=getProbability(typeA, dpInteractionLattice[ x0Abs  + y0POne + z0MOne ]);
            prop/=getProbability(typeA, dpInteractionLattice[ x0Abs  + y0POne + z0PTwo ]);
            prop/=getProbability(typeA, dpInteractionLattice[ x0POne + y0POne + z0MOne ]);
            prop/=getProbability(typeA, dpInteractionLattice[ x0POne + y0POne + z0PTwo ]);
            prop/=getProbability(typeA, dpInteractionLattice[ x0PTwo + y0POne + z0Abs  ]);
            prop/=getProbability(typeA, dpInteractionLattice[ x0PTwo + y0POne + z0POne ]);

            prop/=getProbability(typeA, dpInteractionLattice[ x0Abs  + y0PTwo + z0Abs  ]);
            prop/=getProbability(typeA, dpInteractionLattice[ x0POne + y0PTwo + z0Abs  ]);
            prop/=getProbability(typeA, dpInteractionLattice[ x0Abs  + y0PTwo + z0POne ]);
            prop/=getProbability(typeA, dpInteractionLattice[ x0POne + y0PTwo + z0POne ]);
                
            if(dy==0){prop=1./prop;}                        
            break;
        case 2 : //+-z
            prop*=getProbability(typeA, dpInteractionLattice[ x0Abs  + y0Abs  + z0MTwo ]);
            prop*=getProbability(typeA, dpInteractionLattice[ x0POne + y0Abs  + z0MTwo ]);
            prop*=getProbability(typeA, dpInteractionLattice[ x0Abs  + y0POne + z0MTwo ]);
            prop*=getProbability(typeA, dpInteractionLattice[ x0POne + y0POne + z0MTwo ]);
         
            prop*=getProbability(typeA, dpInteractionLattice[ x0MOne + y0Abs  + z0MOne ]);
            prop*=getProbability(typeA, dpInteractionLattice[ x0MOne + y0POne + z0MOne ]);
            prop*=getProbability(typeA, dpInteractionLattice[ x0Abs  + y0MOne + z0MOne ]);
            prop*=getProbability(typeA, dpInteractionLattice[ x0Abs  + y0PTwo + z0MOne ]);
            prop*=getProbability(typeA, dpInteractionLattice[ x0POne + y0MOne + z0MOne ]);
            prop*=getProbability(typeA, dpInteractionLattice[ x0POne + y0PTwo + z0MOne ]);
            prop*=getProbability(typeA, dpInteractionLattice[ x0PTwo + y0Abs  + z0MOne ]);
            prop*=getProbability(typeA, dpInteractionLattice[ x0PTwo + y0POne + z0MOne ]);

            prop/=getProbability(typeA, dpInteractionLattice[ x0MOne + y0Abs  + z0POne ]);
            prop/=getProbability(typeA, dpInteractionLattice[ x0MOne + y0POne + z0POne ]);
            prop/=getProbability(typeA, dpInteractionLattice[ x0Abs  + y0MOne + z0POne ]);
            prop/=getProbability(typeA, dpInteractionLattice[ x0Abs  + y0PTwo + z0POne ]);
            prop/=getProbability(typeA, dpInteractionLattice[ x0POne + y0MOne + z0POne ]);
            prop/=getProbability(typeA, dpInteractionLattice[ x0POne + y0PTwo + z0POne ]);
            prop/=getProbability(typeA, dpInteractionLattice[ x0PTwo + y0Abs  + z0POne ]);
            prop/=getProbability(typeA, dpInteractionLattice[ x0PTwo + y0POne + z0POne ]);

            prop/=getProbability(typeA, dpInteractionLattice[ x0Abs  + y0Abs  + z0PTwo ]);
            prop/=getProbability(typeA, dpInteractionLattice[ x0POne + y0Abs  + z0PTwo ]);
            prop/=getProbability(typeA, dpInteractionLattice[ x0Abs  + y0POne + z0PTwo ]);
            prop/=getProbability(typeA, dpInteractionLattice[ x0POne + y0POne + z0PTwo ]);
        
            if(dz==0){prop=1./prop;}
            break;
        //TODO : Add diagonal moves 
    }
    return prop;
}
/**
 * @brief add interaction to the species movements
 * 
 * @tparam T_UCoordinateCuda 
 * @param dpPolymerSystem 
 * @param dpPolymerFlags 
 * @param iOffset 
 * @param nMonomers 
 * @param rSeed 
 * @param rGlobalIteration 
 * @param met 
 * @return void 
 */
 template< typename T_UCoordinateCuda >
 __global__ void kernelSimulationScBFMCheckSpeciesInteraction
 (
    T_InteractionTag   * const __restrict__ dpInteractionLattice,
     typename CudaVec4< T_UCoordinateCuda >::value_type
                 const * const __restrict__ dpPolymerSystem         ,
     T_Flags           * const              dpPolymerFlags          ,
     uint32_t            const              iOffset                 ,
     T_Id                const              nMonomers               ,
     uint64_t            const              rSeed                   ,
     uint64_t            const              rGlobalIteration        ,
     Method              const              met
 ){
    for ( T_Id id = blockIdx.x * blockDim.x + threadIdx.x;
        id < nMonomers; id += gridDim.x * blockDim.x ){
        auto const properties = dpPolymerFlags[ id ];
        if ( ( properties & T_Flags(32) ) == T_Flags(0) ) // impossible move
            continue;

        auto direction = properties & T_Flags(31); // 7=0b111 31=0b11111
        auto const r0 = dpPolymerSystem[ id ];
        auto const intProp(calcInteractionProbability( dpInteractionLattice, r0.x, r0.y, r0.z, direction, met ));
        // printf("intProp %d %.15f\n",id, intProp);
        // if ( ! ( rng.rng_d() < intProp  ) ) {
        Saru rng(rGlobalIteration,id+iOffset,rSeed);
        if ( rng.rng_d() < intProp ) {
             /* move is not allowed due to the interaction  */
            // direction ^= T_Flags(32) /* cannot -move-modification */;
            // dpPolymerFlags[ id ] = direction;
            direction += T_Flags(32);
        }
        dpPolymerFlags[ id ] = direction;
     }
 }

 template< typename T_UCoordinateCuda> 
 void UpdaterGPU_Interaction< T_UCoordinateCuda >::launch_CheckSpeciesInteraction(
    const size_t nBlocks, const size_t nThreads, 
    const size_t iSpecies, const uint64_t seed)
 {
    kernelSimulationScBFMCheckSpeciesInteraction< T_UCoordinateCuda > 
    <<< nBlocks, nThreads, 0, mStream >>>(     
        mLatticeInteractionTag->gpu,           
        mPolymerSystemSorted->gpu + mviSubGroupOffsets[ iSpecies ],                                     
        mPolymerFlags->gpu + mviSubGroupOffsets[ iSpecies ],           
        mviSubGroupOffsets[ iSpecies ],                                
        mnElementsInGroup[ iSpecies ],                                 
        seed, 
        hGlobalIterator,                                         
        met
    );
   hGlobalIterator++;
 }

template< typename T_UCoordinateCuda >
__global__ void kernelApplyInteraction(
T_InteractionTag  * const __restrict__ dpInteractionLattice    ,
typename CudaVec4< T_UCoordinateCuda >::value_type
            const * const __restrict__ dpPolymerSystem         ,
T_Flags           * const              dpPolymerFlags          ,
T_Id                const              nMonomers               ,
Method              const              met
){
    for ( T_Id id = blockIdx.x * blockDim.x + threadIdx.x;
        id < nMonomers; id += gridDim.x * blockDim.x ){
        auto const properties = dpPolymerFlags[ id ];
        // if ( ( properties & T_Flags(32) ) == T_Flags(0) ) // impossible move
        if ( ! ( properties & T_Flags(32) ) ) // impossible move
            continue; 
        auto const direction = properties & T_Flags(31); // 7=0b111 31=0b11111
        /** The positions are already updated!
         * Therfore, we substract the direction to obtain the old position,
         * which were assumed in the switch-statement. 
         * Problem : The DXTable_d is not set within this file scope!!!
         * Solution: Rewrite the adressing of the lattice...
         */
        auto const r0 = dpPolymerSystem[ id ] ;
        uint32_t x=r0.x;
        uint32_t y=r0.y;
        uint32_t z=r0.z;

        auto const x0MOne = met.getCurve().linearizeBoxVectorIndexX( x - uint32_t(1) );
        auto const x0Abs  = met.getCurve().linearizeBoxVectorIndexX( x               );
        auto const x0POne = met.getCurve().linearizeBoxVectorIndexX( x + uint32_t(1) );
        auto const x0PTwo = met.getCurve().linearizeBoxVectorIndexX( x + uint32_t(2) );

        auto const y0MOne = met.getCurve().linearizeBoxVectorIndexY( y - uint32_t(1) );
        auto const y0Abs  = met.getCurve().linearizeBoxVectorIndexY( y               );
        auto const y0POne = met.getCurve().linearizeBoxVectorIndexY( y + uint32_t(1) );
        auto const y0PTwo = met.getCurve().linearizeBoxVectorIndexY( y + uint32_t(2) );
    
        auto const z0MOne = met.getCurve().linearizeBoxVectorIndexZ( z - uint32_t(1) );
        auto const z0Abs  = met.getCurve().linearizeBoxVectorIndexZ( z               );
        auto const z0POne = met.getCurve().linearizeBoxVectorIndexZ( z + uint32_t(1) );
        auto const z0PTwo = met.getCurve().linearizeBoxVectorIndexZ( z + uint32_t(2) );
        T_InteractionTag nnTag2(T_InteractionTag(0));
        switch(direction){ 
            case 0:{ //-x
                T_InteractionTag nnTag1(dpInteractionLattice[ x0PTwo + y0Abs + z0Abs  ]);
                if ( 
                    dpInteractionLattice[ x0Abs + y0Abs  + z0Abs  ] != nnTag2 || 
                    dpInteractionLattice[ x0Abs + y0POne + z0Abs  ] != nnTag2 || 
                    dpInteractionLattice[ x0Abs + y0Abs  + z0POne ] != nnTag2 || 
                    dpInteractionLattice[ x0Abs + y0POne + z0POne ] != nnTag2  
                ){
                    printf("Wrong occupation in -x t1: %d %d %d %d at (%d,%d,%d),(%d,%d,%d),(%d,%d,%d),(%d,%d,%d) id=%d\n",
                        static_cast<uint32_t>(dpInteractionLattice[ x0Abs + y0Abs  + z0Abs  ]),
                        static_cast<uint32_t>(dpInteractionLattice[ x0Abs + y0POne + z0Abs  ]),
                        static_cast<uint32_t>(dpInteractionLattice[ x0Abs + y0Abs  + z0POne ]),
                        static_cast<uint32_t>(dpInteractionLattice[ x0Abs + y0POne + z0POne ]),
                        uint32_t(r0.x), uint32_t(r0.y)  , uint32_t(r0.z)   ,
                        uint32_t(r0.x), uint32_t(r0.y)+1, uint32_t(r0.z)   ,
                        uint32_t(r0.x), uint32_t(r0.y)  , uint32_t(r0.z)+1 ,
                        uint32_t(r0.x), uint32_t(r0.y)+1, uint32_t(r0.z)+1 ,id
                    );
                }
                if ( 
                    dpInteractionLattice[ x0PTwo + y0Abs  + z0Abs  ] != nnTag1 || 
                    dpInteractionLattice[ x0PTwo + y0POne + z0Abs  ] != nnTag1 || 
                    dpInteractionLattice[ x0PTwo + y0Abs  + z0POne ] != nnTag1 || 
                    dpInteractionLattice[ x0PTwo + y0POne + z0POne ] != nnTag1  
                ){
                    printf("Wrong occupation in -x t2: %d %d %d %d at (%d,%d,%d),(%d,%d,%d),(%d,%d,%d),(%d,%d,%d) id=%d\n",
                        static_cast<uint32_t>(dpInteractionLattice[ x0PTwo + y0Abs  + z0Abs  ]),
                        static_cast<uint32_t>(dpInteractionLattice[ x0PTwo + y0POne + z0Abs  ]),
                        static_cast<uint32_t>(dpInteractionLattice[ x0PTwo + y0Abs  + z0POne ]),
                        static_cast<uint32_t>(dpInteractionLattice[ x0PTwo + y0POne + z0POne ]),
                        uint32_t(r0.x)+2, uint32_t(r0.y)  , uint32_t(r0.z)   ,
                        uint32_t(r0.x)+2, uint32_t(r0.y)+1, uint32_t(r0.z)   ,
                        uint32_t(r0.x)+2, uint32_t(r0.y)  , uint32_t(r0.z)+1 ,
                        uint32_t(r0.x)+2, uint32_t(r0.y)+1, uint32_t(r0.z)+1 ,id
                    );
                }

                dpInteractionLattice[ x0Abs + y0Abs  + z0Abs  ] = nnTag1;
                dpInteractionLattice[ x0Abs + y0POne + z0Abs  ] = nnTag1;
                dpInteractionLattice[ x0Abs + y0Abs  + z0POne ] = nnTag1;
                dpInteractionLattice[ x0Abs + y0POne + z0POne ] = nnTag1;
                
                dpInteractionLattice[ x0PTwo + y0Abs  + z0Abs  ] = nnTag2;
                dpInteractionLattice[ x0PTwo + y0POne + z0Abs  ] = nnTag2;
                dpInteractionLattice[ x0PTwo + y0Abs  + z0POne ] = nnTag2;
                dpInteractionLattice[ x0PTwo + y0POne + z0POne ] = nnTag2;
                }
                break;
            case 1:{ //+x
                T_InteractionTag nnTag1(dpInteractionLattice[ x0MOne + y0Abs + z0Abs  ]);
                if ( 
                    dpInteractionLattice[ x0POne + y0Abs  + z0Abs  ] != nnTag2 || 
                    dpInteractionLattice[ x0POne + y0POne + z0Abs  ] != nnTag2 || 
                    dpInteractionLattice[ x0POne + y0Abs  + z0POne ] != nnTag2 || 
                    dpInteractionLattice[ x0POne + y0POne + z0POne ] != nnTag2  
                ){
                    printf("Wrong occupation in +x t1: %d %d %d %d at (%d,%d,%d),(%d,%d,%d),(%d,%d,%d),(%d,%d,%d) id=%d\n",
                        static_cast<uint32_t>(dpInteractionLattice[ x0POne + y0Abs  + z0Abs  ]),
                        static_cast<uint32_t>(dpInteractionLattice[ x0POne + y0POne + z0Abs  ]),
                        static_cast<uint32_t>(dpInteractionLattice[ x0POne + y0Abs  + z0POne ]),
                        static_cast<uint32_t>(dpInteractionLattice[ x0POne + y0POne + z0POne ]),
                        uint32_t(r0.x)+1, uint32_t(r0.y)  , uint32_t(r0.z)   ,
                        uint32_t(r0.x)+1, uint32_t(r0.y)+1, uint32_t(r0.z)   ,
                        uint32_t(r0.x)+1, uint32_t(r0.y)  , uint32_t(r0.z)+1 ,
                        uint32_t(r0.x)+1, uint32_t(r0.y)+1, uint32_t(r0.z)+1 ,id
                    );
                }
                if ( 
                    dpInteractionLattice[ x0MOne + y0Abs  + z0Abs  ] != nnTag1 || 
                    dpInteractionLattice[ x0MOne + y0POne + z0Abs  ] != nnTag1 || 
                    dpInteractionLattice[ x0MOne + y0Abs  + z0POne ] != nnTag1 || 
                    dpInteractionLattice[ x0MOne + y0POne + z0POne ] != nnTag1  
                ){
                    printf("Wrong occupation in +x t2: %d %d %d %d at (%d,%d,%d),(%d,%d,%d),(%d,%d,%d),(%d,%d,%d) id=%d\n",
                        static_cast<uint32_t>(dpInteractionLattice[ x0MOne + y0Abs  + z0Abs  ]),
                        static_cast<uint32_t>(dpInteractionLattice[ x0MOne + y0POne + z0Abs   ]),
                        static_cast<uint32_t>(dpInteractionLattice[ x0MOne + y0Abs  + z0POne  ]),
                        static_cast<uint32_t>(dpInteractionLattice[ x0MOne + y0POne + z0POne]),
                        uint32_t(r0.x)-1, uint32_t(r0.y)  , uint32_t(r0.z)   ,
                        uint32_t(r0.x)-1, uint32_t(r0.y)+1, uint32_t(r0.z)   ,
                        uint32_t(r0.x)-1, uint32_t(r0.y)  , uint32_t(r0.z)+1 ,
                        uint32_t(r0.x)-1, uint32_t(r0.y)+1, uint32_t(r0.z)+1 ,id
                    );
                }
                dpInteractionLattice[ x0POne + y0Abs  + z0Abs  ] = nnTag1;
                dpInteractionLattice[ x0POne + y0POne + z0Abs  ] = nnTag1;
                dpInteractionLattice[ x0POne + y0Abs  + z0POne ] = nnTag1;
                dpInteractionLattice[ x0POne + y0POne + z0POne ] = nnTag1;

                dpInteractionLattice[ x0MOne  + y0Abs  + z0Abs  ] = nnTag2;
                dpInteractionLattice[ x0MOne  + y0POne + z0Abs  ] = nnTag2;
                dpInteractionLattice[ x0MOne  + y0Abs  + z0POne ] = nnTag2;
                dpInteractionLattice[ x0MOne  + y0POne + z0POne ] = nnTag2;
                }

                break;
            case 2:{ //-y
                T_InteractionTag nnTag1(dpInteractionLattice[ x0Abs + y0PTwo + z0Abs  ]);
                if ( 
                    dpInteractionLattice[ x0Abs  + y0Abs + z0Abs  ] != nnTag2 || 
                    dpInteractionLattice[ x0POne + y0Abs + z0Abs  ] != nnTag2 || 
                    dpInteractionLattice[ x0Abs  + y0Abs + z0POne ] != nnTag2 || 
                    dpInteractionLattice[ x0POne + y0Abs + z0POne ] != nnTag2  
                ){
                    printf("Wrong occupation in -y t1: %d %d %d %d at (%d,%d,%d),(%d,%d,%d),(%d,%d,%d),(%d,%d,%d) id=%d\n",
                        static_cast<uint32_t>(dpInteractionLattice[ x0Abs  + y0Abs + z0Abs  ]),
                        static_cast<uint32_t>(dpInteractionLattice[ x0POne + y0Abs + z0Abs  ]),
                        static_cast<uint32_t>(dpInteractionLattice[ x0Abs  + y0Abs + z0POne ]),
                        static_cast<uint32_t>(dpInteractionLattice[ x0POne + y0Abs + z0POne ]),
                        uint32_t(r0.x)  , uint32_t(r0.y)  , uint32_t(r0.z)   ,
                        uint32_t(r0.x)+1, uint32_t(r0.y)  , uint32_t(r0.z)   ,
                        uint32_t(r0.x)  , uint32_t(r0.y)  , uint32_t(r0.z)+1 ,
                        uint32_t(r0.x)+1, uint32_t(r0.y)  , uint32_t(r0.z)+1 ,id
                    );
                }
                if ( 
                    dpInteractionLattice[ x0Abs  + y0PTwo + z0Abs  ] != nnTag1 || 
                    dpInteractionLattice[ x0POne + y0PTwo + z0Abs  ] != nnTag1 || 
                    dpInteractionLattice[ x0Abs  + y0PTwo + z0POne ] != nnTag1 || 
                    dpInteractionLattice[ x0POne + y0PTwo + z0POne ] != nnTag1  
                ){
                    printf("Wrong occupation in -y t2: %d %d %d %d at (%d,%d,%d),(%d,%d,%d),(%d,%d,%d),(%d,%d,%d) id=%d\n",
                        static_cast<uint32_t>(dpInteractionLattice[ x0Abs  + y0PTwo + z0Abs  ]),
                        static_cast<uint32_t>(dpInteractionLattice[ x0POne + y0PTwo + z0Abs  ]),
                        static_cast<uint32_t>(dpInteractionLattice[ x0Abs  + y0PTwo + z0POne ]),
                        static_cast<uint32_t>(dpInteractionLattice[ x0POne + y0PTwo + z0POne ]),
                        uint32_t(r0.x)  , uint32_t(r0.y)+2, uint32_t(r0.z)   ,
                        uint32_t(r0.x)+1, uint32_t(r0.y)+2, uint32_t(r0.z)   ,
                        uint32_t(r0.x)  , uint32_t(r0.y)+2, uint32_t(r0.z)+1 ,
                        uint32_t(r0.x)+1, uint32_t(r0.y)+2, uint32_t(r0.z)+1 ,id
                    );
                } 

                dpInteractionLattice[ x0Abs  + y0Abs + z0Abs  ] = nnTag1;
                dpInteractionLattice[ x0POne + y0Abs + z0Abs  ] = nnTag1;
                dpInteractionLattice[ x0Abs  + y0Abs + z0POne ] = nnTag1;
                dpInteractionLattice[ x0POne + y0Abs + z0POne ] = nnTag1;
                
                dpInteractionLattice[ x0Abs  + y0PTwo + z0Abs  ] = nnTag2;
                dpInteractionLattice[ x0POne + y0PTwo + z0Abs  ] = nnTag2;
                dpInteractionLattice[ x0Abs  + y0PTwo + z0POne ] = nnTag2;
                dpInteractionLattice[ x0POne + y0PTwo + z0POne ] = nnTag2;
                }
                break;
            case 3:{ //+y
                T_InteractionTag nnTag1(dpInteractionLattice[ x0Abs + y0MOne + z0Abs  ]);
                if ( 
                    dpInteractionLattice[ x0Abs  + y0POne + z0Abs  ] != nnTag2 || 
                    dpInteractionLattice[ x0POne + y0POne + z0Abs  ] != nnTag2 || 
                    dpInteractionLattice[ x0Abs  + y0POne + z0POne ] != nnTag2 || 
                    dpInteractionLattice[ x0POne + y0POne + z0POne ] != nnTag2  
                ){
                    printf("Wrong occupation in +y t1: %d %d %d %d at (%d,%d,%d),(%d,%d,%d),(%d,%d,%d),(%d,%d,%d) id=%d\n",
                        static_cast<uint32_t>(dpInteractionLattice[ x0Abs  + y0POne + z0Abs  ]),
                        static_cast<uint32_t>(dpInteractionLattice[ x0POne + y0POne + z0Abs  ]),
                        static_cast<uint32_t>(dpInteractionLattice[ x0Abs  + y0POne + z0POne ]),
                        static_cast<uint32_t>(dpInteractionLattice[ x0POne + y0POne + z0POne ]),
                        uint32_t(r0.x)  , uint32_t(r0.y)+1, uint32_t(r0.z)   ,
                        uint32_t(r0.x)+1, uint32_t(r0.y)+1, uint32_t(r0.z)   ,
                        uint32_t(r0.x)  , uint32_t(r0.y)+1, uint32_t(r0.z)+1 ,
                        uint32_t(r0.x)+1, uint32_t(r0.y)+1, uint32_t(r0.z)+1 ,id
                    );
                }
                if ( 
                    dpInteractionLattice[ x0Abs  + y0MOne + z0Abs  ] != nnTag1 || 
                    dpInteractionLattice[ x0POne + y0MOne + z0Abs  ] != nnTag1 || 
                    dpInteractionLattice[ x0Abs  + y0MOne + z0POne ] != nnTag1 || 
                    dpInteractionLattice[ x0POne + y0MOne + z0POne ] != nnTag1  
                ){
                    printf("Wrong occupation in +y t2: %d %d %d %d at (%d,%d,%d),(%d,%d,%d),(%d,%d,%d),(%d,%d,%d) id=%d\n",
                        static_cast<uint32_t>(dpInteractionLattice[ x0Abs  + y0MOne + z0Abs  ]),
                        static_cast<uint32_t>(dpInteractionLattice[ x0POne + y0MOne + z0Abs  ]),
                        static_cast<uint32_t>(dpInteractionLattice[ x0Abs  + y0MOne + z0POne ]),
                        static_cast<uint32_t>(dpInteractionLattice[ x0POne + y0MOne + z0POne ]),
                        uint32_t(r0.x)  , uint32_t(r0.y)-1, uint32_t(r0.z)   ,
                        uint32_t(r0.x)+1, uint32_t(r0.y)-1, uint32_t(r0.z)   ,
                        uint32_t(r0.x)  , uint32_t(r0.y)-1, uint32_t(r0.z)+1 ,
                        uint32_t(r0.x)+1, uint32_t(r0.y)-1, uint32_t(r0.z)+1 ,id
                    );
                }
                dpInteractionLattice[ x0Abs  + y0POne + z0Abs  ] = nnTag1;
                dpInteractionLattice[ x0POne + y0POne + z0Abs  ] = nnTag1;
                dpInteractionLattice[ x0Abs  + y0POne + z0POne ] = nnTag1;
                dpInteractionLattice[ x0POne + y0POne + z0POne ] = nnTag1;

                dpInteractionLattice[ x0Abs  + y0MOne  + z0Abs  ] = nnTag2;
                dpInteractionLattice[ x0POne + y0MOne  + z0Abs  ] = nnTag2;
                dpInteractionLattice[ x0Abs  + y0MOne  + z0POne ] = nnTag2;
                dpInteractionLattice[ x0POne + y0MOne  + z0POne ] = nnTag2;
                }
                break;
            case 4:{ //-z
                T_InteractionTag nnTag1(dpInteractionLattice[ x0Abs + y0Abs + z0PTwo  ]);
                if ( 
                    dpInteractionLattice[ x0Abs  + y0Abs  + z0Abs ] != nnTag2 || 
                    dpInteractionLattice[ x0Abs  + y0POne + z0Abs ] != nnTag2 || 
                    dpInteractionLattice[ x0POne + y0Abs  + z0Abs ] != nnTag2 || 
                    dpInteractionLattice[ x0POne + y0POne + z0Abs ] != nnTag2  
                ){
                    printf("Wrong occupation in -z t1: %d %d %d %d at (%d,%d,%d),(%d,%d,%d),(%d,%d,%d),(%d,%d,%d) id=%d\n",
                        static_cast<uint32_t>(dpInteractionLattice[ x0Abs  + y0Abs  + z0Abs ]),
                        static_cast<uint32_t>(dpInteractionLattice[ x0Abs  + y0POne + z0Abs ]),
                        static_cast<uint32_t>(dpInteractionLattice[ x0POne + y0Abs  + z0Abs ]),
                        static_cast<uint32_t>(dpInteractionLattice[ x0POne + y0POne + z0Abs ]),
                        uint32_t(r0.x)  , uint32_t(r0.y)  , uint32_t(r0.z)   ,
                        uint32_t(r0.x)  , uint32_t(r0.y)+1, uint32_t(r0.z)   ,
                        uint32_t(r0.x)+1, uint32_t(r0.y)  , uint32_t(r0.z)   ,
                        uint32_t(r0.x)+1, uint32_t(r0.y)+1, uint32_t(r0.z)  ,id
                    );
                }
                if ( 
                    dpInteractionLattice[ x0Abs  + y0Abs  + z0PTwo ] != nnTag1 || 
                    dpInteractionLattice[ x0Abs  + y0POne + z0PTwo ] != nnTag1 || 
                    dpInteractionLattice[ x0POne + y0Abs  + z0PTwo ] != nnTag1 || 
                    dpInteractionLattice[ x0POne + y0POne + z0PTwo ] != nnTag1  
                ){
                    printf("Wrong occupation in -z t2: %d %d %d %d at (%d,%d,%d),(%d,%d,%d),(%d,%d,%d),(%d,%d,%d) id=%d\n",
                        static_cast<uint32_t>(dpInteractionLattice[ x0Abs  + y0Abs  + z0PTwo ]),
                        static_cast<uint32_t>(dpInteractionLattice[ x0Abs  + y0POne + z0PTwo ]),
                        static_cast<uint32_t>(dpInteractionLattice[ x0POne + y0Abs  + z0PTwo ]),
                        static_cast<uint32_t>(dpInteractionLattice[ x0POne + y0POne + z0PTwo ]),
                        uint32_t(r0.x)  , uint32_t(r0.y)  , uint32_t(r0.z)+2 ,
                        uint32_t(r0.x)  , uint32_t(r0.y)+1, uint32_t(r0.z)+2 ,
                        uint32_t(r0.x)+1, uint32_t(r0.y)  , uint32_t(r0.z)+2 ,
                        uint32_t(r0.x)+1, uint32_t(r0.y)+1, uint32_t(r0.z)+2 ,id
                    );
                }
                dpInteractionLattice[ x0Abs  + y0Abs  + z0Abs ] = nnTag1;
                dpInteractionLattice[ x0Abs  + y0POne + z0Abs ] = nnTag1;
                dpInteractionLattice[ x0POne + y0Abs  + z0Abs ] = nnTag1;
                dpInteractionLattice[ x0POne + y0POne + z0Abs ] = nnTag1;
                
                dpInteractionLattice[ x0Abs  + y0Abs  + z0PTwo ] = nnTag2;
                dpInteractionLattice[ x0Abs  + y0POne + z0PTwo ] = nnTag2;
                dpInteractionLattice[ x0POne + y0Abs  + z0PTwo ] = nnTag2;
                dpInteractionLattice[ x0POne + y0POne + z0PTwo ] = nnTag2;
                }

                break;
            case 5:{ //+z
                T_InteractionTag nnTag1(dpInteractionLattice[ x0Abs + y0Abs + z0MOne  ]);
                if ( 
                    dpInteractionLattice[ x0Abs  + y0Abs  + z0POne ] != nnTag2 || 
                    dpInteractionLattice[ x0Abs  + y0POne + z0POne ] != nnTag2 || 
                    dpInteractionLattice[ x0POne + y0Abs  + z0POne ] != nnTag2 || 
                    dpInteractionLattice[ x0POne + y0POne + z0POne ] != nnTag2  
                ){
                    printf("Wrong occupation in +z t1: %d %d %d %d at (%d,%d,%d),(%d,%d,%d),(%d,%d,%d),(%d,%d,%d) id=%d\n",
                        static_cast<uint32_t>(dpInteractionLattice[ x0Abs  + y0Abs  + z0POne ]),
                        static_cast<uint32_t>(dpInteractionLattice[ x0Abs  + y0POne + z0POne ]),
                        static_cast<uint32_t>(dpInteractionLattice[ x0POne + y0Abs  + z0POne ]),
                        static_cast<uint32_t>(dpInteractionLattice[ x0POne + y0POne + z0POne ]),
                        uint32_t(r0.x)  , uint32_t(r0.y)  , uint32_t(r0.z)+1 ,
                        uint32_t(r0.x)  , uint32_t(r0.y)+1, uint32_t(r0.z)+1 ,
                        uint32_t(r0.x)+1, uint32_t(r0.y)  , uint32_t(r0.z)+1 ,
                        uint32_t(r0.x)+1, uint32_t(r0.y)+1, uint32_t(r0.z)+1 ,id
                    );
                }
                if ( 
                    dpInteractionLattice[ x0Abs  + y0Abs  + z0MOne ] != nnTag1 || 
                    dpInteractionLattice[ x0Abs  + y0POne + z0MOne ] != nnTag1 || 
                    dpInteractionLattice[ x0POne + y0Abs  + z0MOne ] != nnTag1 || 
                    dpInteractionLattice[ x0POne + y0POne + z0MOne ] != nnTag1  
                ){
                    printf("Wrong occupation in +z t2 : %d %d %d %d at (%d,%d,%d),(%d,%d,%d),(%d,%d,%d),(%d,%d,%d) id=%d\n",
                        static_cast<uint32_t>(dpInteractionLattice[ x0Abs  + y0Abs  + z0MOne ]),
                        static_cast<uint32_t>(dpInteractionLattice[ x0Abs  + y0POne + z0MOne ]),
                        static_cast<uint32_t>(dpInteractionLattice[ x0POne + y0Abs  + z0MOne ]),
                        static_cast<uint32_t>(dpInteractionLattice[ x0POne + y0POne + z0MOne ]),
                        uint32_t(r0.x)  , uint32_t(r0.y)  , uint32_t(r0.z)-1 ,
                        uint32_t(r0.x)  , uint32_t(r0.y)+1, uint32_t(r0.z)-1 ,
                        uint32_t(r0.x)+1, uint32_t(r0.y)  , uint32_t(r0.z)-1 ,
                        uint32_t(r0.x)+1, uint32_t(r0.y)+1, uint32_t(r0.z)-1 ,id
                    );
                }
                dpInteractionLattice[ x0Abs  + y0Abs  + z0POne ] = nnTag1;
                dpInteractionLattice[ x0Abs  + y0POne + z0POne ] = nnTag1;
                dpInteractionLattice[ x0POne + y0Abs  + z0POne ] = nnTag1;
                dpInteractionLattice[ x0POne + y0POne + z0POne ] = nnTag1;

                dpInteractionLattice[ x0Abs  + y0Abs  + z0MOne  ] = nnTag2;
                dpInteractionLattice[ x0Abs  + y0POne + z0MOne  ] = nnTag2;
                dpInteractionLattice[ x0POne + y0Abs  + z0MOne  ] = nnTag2;
                dpInteractionLattice[ x0POne + y0POne + z0MOne  ] = nnTag2;
                }
                break;
        }
    }
}
template< typename T_UCoordinateCuda >
void UpdaterGPU_Interaction< T_UCoordinateCuda >::launch_ApplyInteraction(
  const size_t nBlocks , const size_t   nThreads, const size_t iSpecies
){ 
	kernelApplyInteraction<T_UCoordinateCuda><<<nBlocks,nThreads,0,mStream>>>(
        mLatticeInteractionTag->gpu,           
        mPolymerSystemSorted->gpu + mviSubGroupOffsets[ iSpecies ],                                     
        mPolymerFlags->gpu + mviSubGroupOffsets[ iSpecies ],                                         
        mnElementsInGroup[ iSpecies ],                           
        met
	);
}
///////////////////////////////////////////////////////////////////////////////
//Define othe member functions/////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////
template< typename T_UCoordinateCuda > 
UpdaterGPU_Interaction<T_UCoordinateCuda>::UpdaterGPU_Interaction():
BaseClass 				(	   ),         
mLatticeInteractionTag  ( NULL ),
mInteractionTag        	( NULL )
{
    /**
     * Log control.
     * Note that "Check" controls not the output, but the actualy checks
     * If a checks needs to always be done, then do that check and declare
     * the output as "Info" log level
     */
    mLog.file( __FILENAME__ );
    mLog.activate( "Check"     );
    mLog.activate( "Error"     );
    mLog.activate( "Info"      );
    mLog.deactivate( "Stats"     );
    mLog.deactivate( "Warning"   );
    for(size_t n=0;n<maxInteractionType;n++){
      	for(size_t m=0;m<maxInteractionType;m++){	
			interactionTable[m][n]=0.0;
			probabilityLookup[m][n]=1.0;
        }
    }
};
template< typename T_UCoordinateCuda > 
void UpdaterGPU_Interaction<T_UCoordinateCuda>::destruct(){
	DeleteMirroredObject deletePointer;
	deletePointer( mLatticeInteractionTag, "mLatticeInteractionTag");
    deletePointer(        mInteractionTag,        "mInteractionTag");
	if ( deletePointer.nBytesFreed > 0 ){
		mLog( "Info" )
			<< "Freed a total of "
			<< prettyPrintBytes( deletePointer.nBytesFreed )
			<< " on GPU and host RAM.\n";
	}
}
template< typename T_UCoordinateCuda > 
UpdaterGPU_Interaction<T_UCoordinateCuda>::~UpdaterGPU_Interaction(){
	this->destruct();    
	destruct();
}
template< typename T_UCoordinateCuda >
void UpdaterGPU_Interaction<T_UCoordinateCuda>::cleanup(){
    this->destruct();    
    destruct();
    hipDeviceSynchronize();
    hipProfilerStop();
}
template < typename T_UCoordinateCuda >
void UpdaterGPU_Interaction<T_UCoordinateCuda>::initialize(){
	BaseClass::setAutoColoring(false);
    mLog( "Info" )<< "Start manual coloring of the graph...\n" ;
    bool const bUniformColors = true; // setting this to true should yield more performance as the kernels are uniformly utilized
    //map with: key=interactionTag, values=number of Monomers with interaction TAg
    std::map<uint32_t,std::vector<uint32_t> > newToOldNNIDs; 
    std::vector<uint32_t> oldToNewNNIDs(mnAllMonomers,0); 
    for(auto i=0; i < mnAllMonomers; i++){
        newToOldNNIDs[getAttributeTag(i)].push_back(i);
        oldToNewNNIDs[i]=newToOldNNIDs[getAttributeTag(i)].size()-1;
    }
    for(auto i=0; i < 20; i++)
        mLog( "Info" )<< "oldToNewNNIDs["<<i<<"]="<< oldToNewNNIDs[i]<<"\n" ; 
    //vector with the interaction Tags
    std::vector<uint32_t> interactionTags; 
    //offset of the number of monomers with the interaction tag 
    std::vector<uint32_t> interactionTagsOffset; 
    //inteaction tag sorted 
    //interaction Tag are for example : 2 3 7 , which are sorted to 0 1 2
    std::map<uint32_t,uint32_t> interactionTagSorted;
    uint32_t tmpCounter(0);
    for(auto it=newToOldNNIDs.begin(); it!=newToOldNNIDs.end();it++){
        interactionTags.push_back(it->first );
        interactionTagSorted[it->first]=tmpCounter;
        tmpCounter++;
    }
    interactionTagsOffset.push_back(0);
    for(auto i=1; i< interactionTags.size();i++){
        interactionTagsOffset.push_back(interactionTagsOffset[i-1] + newToOldNNIDs.at(interactionTags[i]).size());
    }

    mLog( "Info" )<< "There are "<< interactionTags.size()<<" interaction species.\n" ;
    for (auto i=0; i < interactionTags.size(); i++)
        mLog( "Info" )<< "interaction species type "<< interactionTags[i]<<"->"<<interactionTagSorted[interactionTags[i]]<< " size="<<newToOldNNIDs.at(interactionTags[i]).size() << "\n" ;
    //create a neighboring list which contains only on interaction tag species
    std::vector< std::vector< MonomerEdges > >  mSpeciesNeighbors;
    for (auto i=0; i < interactionTags.size(); i++){
        std::vector< MonomerEdges > neighbors(newToOldNNIDs[interactionTags[i]].size(),MonomerEdges());
        mSpeciesNeighbors.push_back(neighbors);
    }
    for(auto i=0; i < mnAllMonomers; i++){
        auto attribute(interactionTagSorted.at(getAttributeTag(i)));
        auto oldID(i);
        MonomerEdges oldNeighbors(mNeighbors->host[oldID]);
        MonomerEdges newNeighbors;
        newNeighbors.size=0;
        for(auto j=0;j<oldNeighbors.size;j++){
            if (getAttributeTag(oldNeighbors.neighborIds[j]) != getAttributeTag(i) ) continue;
            auto neighborID( oldToNewNNIDs[ oldNeighbors.neighborIds[j] ] );            
            newNeighbors.neighborIds[newNeighbors.size]=neighborID;
            newNeighbors.size++;
        }
        auto newID(oldToNewNNIDs[oldID]);
        if (i <20 )
            std::cout << oldID << " " << newID <<  " " << attribute<<std::endl;
        mSpeciesNeighbors[attribute][newID]=newNeighbors;
    }
    // use the automatic coloring algorithm within one interaction tag species
    std::vector< std::vector< T_Color > > mSpeciesGroupIds;
    for (auto i=0; i < interactionTags.size(); i++){
        mSpeciesGroupIds.push_back(
            graphColoring< std::vector<MonomerEdges> const, T_Id, T_Color >(
                mSpeciesNeighbors[i], 
                newToOldNNIDs.at(interactionTags.at(i)).size(), 
                bUniformColors,
                []( std::vector<MonomerEdges> const & x, T_Id const & i ){ return x[i].size; },
                []( std::vector<MonomerEdges> const & x, T_Id const & i, size_t const & j ){ return x[i].neighborIds[j]; }
            )
        );
    }
    //resort the colors to the initial ids 
    mGroupIds.resize(mnAllMonomers,0);
    // for (auto i=0; i < mnAllMonomers; i++){
    auto colorOffset(0);
    for (auto i=0; i < mSpeciesGroupIds.size(); i++){
        auto attribute(interactionTags[i]);
        std::map<uint32_t,uint32_t> usedColors;
        for(auto j=0; j < mSpeciesGroupIds[i].size(); j++){            
            auto oldID(newToOldNNIDs[attribute][j]);
            mGroupIds[oldID]=mSpeciesGroupIds[i][j]+colorOffset;
            usedColors[mGroupIds[oldID]]++;
        }
        colorOffset+=usedColors.size();
    }
    mLog( "Info" )<< "Colors:\n";
    for(auto i=0; i <20;i++)
		mLog( "Info" )<< "mGroups[" << i << "]= "<< mGroupIds[i] <<"\n" ;

	mLog( "Info" )<< "Start manual coloring of the graph...done\n" ;

	mLog( "Info" )<< "Initialize baseclass \n" ;
	BaseClass::initialize();
    size_t nBytesInteractionTagTmp = mnMonomersPadded* sizeof(T_InteractionTag);
    mLog( "Info" ) << "Allocate "<< nBytesInteractionTagTmp/1024<<"kB  memory for mInteractionTag \n";  
    mInteractionTag  = new MirroredTexture< T_InteractionTag >( nBytesInteractionTagTmp, mStream );
    miToiNew->popAsync();
	CUDA_ERROR( hipStreamSynchronize( mStream ) );
    for( auto i=0;i<mnAllMonomers; i++)
        mInteractionTag->host[miToiNew->host[i]]=static_cast<uint8_t>(getAttributeTag(i)); 
    mInteractionTag->push(0);
    hipStreamSynchronize( mStream );


	{ decltype( dcBoxX  ) x = mBoxX  ; CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dcBoxX  ), &x, sizeof(x) ) ); }
	{ decltype( dcBoxY  ) x = mBoxY  ; CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dcBoxY  ), &x, sizeof(x) ) ); }
	{ decltype( dcBoxZ  ) x = mBoxZ  ; CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dcBoxZ  ), &x, sizeof(x) ) ); }
	{ decltype( dcBoxXM1) x = mBoxXM1; CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dcBoxXM1), &x, sizeof(x) ) ); }
	{ decltype( dcBoxYM1) x = mBoxYM1; CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dcBoxYM1), &x, sizeof(x) ) ); }
	{ decltype( dcBoxZM1) x = mBoxZM1; CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dcBoxZM1), &x, sizeof(x) ) ); }
    uint64_t mBoxXLog2(0), mBoxXYLog2(0);
    { auto dummy = mBoxX ; while ( dummy >>= 1 ) ++mBoxXLog2;
      dummy = mBoxX*mBoxY; while ( dummy >>= 1 ) ++mBoxXYLog2;}
    { decltype( dcBoxXLog2  ) x = mBoxXLog2  ; CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dcBoxXLog2 ), &x, sizeof(x) ) ); }
    { decltype( dcBoxXYLog2 ) x = mBoxXYLog2 ; CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dcBoxXYLog2), &x, sizeof(x) ) ); } 

	uint32_t tmp_DXTableNN[18] = {  0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };
	uint32_t tmp_DYTableNN[18] = {  0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };
	uint32_t tmp_DZTableNN[18] = {  0, 0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };
	CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( DXTableNN_d), tmp_DXTableNN, sizeof( tmp_DXTableNN ) ) ); 
	CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( DYTableNN_d), tmp_DYTableNN, sizeof( tmp_DYTableNN ) ) );
	CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( DZTableNN_d), tmp_DZTableNN, sizeof( tmp_DZTableNN ) ) );
    CheckBoxDimensions<<<1,1,0,mStream>>>();
    CUDA_ERROR( hipStreamSynchronize( mStream ) );
	mLog( "Info" )<< "Initialize baseclass.done. \n" ;	

	initializeInteractionLattice();
    auto const nSpecies = mnElementsInGroup.size();
    for ( uint32_t iSpecies = 0; iSpecies < nSpecies; ++iSpecies ){
        /* randomly choose which monomer group to advance */
        auto const nThreads = 256;
        auto const nBlocks  = ceilDiv( mnElementsInGroup[ iSpecies ], nThreads );
        launch_initializeInteractionLattice(nBlocks,nThreads,iSpecies);
    }
    checkInteractionLatticeOccupation();
	mLog( "Info" )<< "Initialize lattice.done. \n" ;

    for (auto i=0; i<20; i++ )
        for (auto j=0; j<20; j++ )
            mLog( "Info" )<< "interaction: probabilityLookup[" <<  i  <<","<<j << "]="<< probabilityLookup[i+1][j+1]  <<"\n";
    CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL( dcNNProbability), probabilityLookup, sizeof(probabilityLookup) ));
    checkInteractionLatticeOccupation();
    kernelPrintTagType<<<20,20>>>();
    checkCurve<<<32,1,0,mStream>>>(met);
    CUDA_ERROR( hipStreamSynchronize( mStream ) ); // finish e.g. initializations
}
////////////////////////////////////////////////////////////////////////////////
//implement setter function for the interaction tags and their energy //////////
////////////////////////////////////////////////////////////////////////////////
template< typename T_UCoordinateCuda >
void UpdaterGPU_Interaction<T_UCoordinateCuda>::setInteractionTag(
    uint32_t id, uint8_t tag ){
    setAttributeTag(id, static_cast<uint32_t>(tag));
}
////////////////////////////////////////////////////////////////////////////////
template< typename T_UCoordinateCuda >
void UpdaterGPU_Interaction<T_UCoordinateCuda>::setNNInteraction(
    int32_t typeA, int32_t typeB, double energy){
    if(0<typeA && typeA<=maxInteractionType && 0<typeB && typeB<=maxInteractionType){
        interactionTable[typeA+1][typeB+1]=energy;
        interactionTable[typeB+1][typeA+1]=energy;
        probabilityLookup[typeA+1][typeB+1]=exp(energy);
        probabilityLookup[typeB+1][typeA+1]=exp(energy);
        std::cout<<"set interation between types ";
        std::cout<<typeA<<" and "<<typeB<<" to "<<energy<<"kT\n";
    } else {
        std::stringstream errormessage;
        errormessage<<"UpdaterGPU_Interaction::setNNInteraction(typeA,typeB,energy).\n";
        errormessage<<"typeA "<<typeA<<" typeB "<<typeB<<": Types out of range\n";
        throw std::runtime_error(errormessage.str());
    }
}
////////////////////////////////////////////////////////////////////////////////
template< typename T_UCoordinateCuda >
double UpdaterGPU_Interaction<T_UCoordinateCuda>::getNNInteraction(int32_t typeA, 
    int32_t typeB) const {
    if(0<typeA && typeA<=maxInteractionType && 0<typeB && typeB<=maxInteractionType)
        return interactionTable[typeA+1][typeB+1];
    else{
        std::stringstream errormessage;
        errormessage<<"UpdaterGPU_Interaction::getNNInteraction(typeA,typeB).\n";
        errormessage<<"typeA "<<typeA<<" typeB "<<typeB<<": Types out of range\n";
        throw std::runtime_error(errormessage.str());
    }
}
////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
template< typename T_UCoordinateCuda  >
void UpdaterGPU_Interaction< T_UCoordinateCuda >::runSimulationOnGPU( 
    uint32_t const nMonteCarloSteps ){
    std::clock_t const t0 = std::clock();
    CUDA_ERROR( hipStreamSynchronize( mStream ) ); // finish e.g. initializations
    CUDA_ERROR( hipMemcpy( mPolymerSystemSortedOld->gpu, 
                            mPolymerSystemSorted->gpu, 
                            mPolymerSystemSortedOld->nBytes, 
                            hipMemcpyDeviceToDevice ) );
    auto const nSpecies = mnElementsInGroup.size();
    AutomaticThreadChooser chooseThreads(nSpecies);
    chooseThreads.initialize(mCudaProps);
    std::vector< uint64_t > nSpeciesChosen( nSpecies ,0 );
    /* run simulation */
    for ( uint32_t iStep = 0; iStep < nMonteCarloSteps; ++iStep, ++mAge ){
        if ( useOverflowChecks ){
            /**
             * for uint8_t we have to check for overflows every 127 steps, as
             * for 128 steps we couldn't say whether it actually moved 128 steps
             * or whether it moved 128 steps in the other direction and was wrapped
             * to be equal to the hypothetical monomer above
             */
            auto constexpr boxSizeCudaType = 1ll << ( sizeof( T_UCoordinateCuda ) * CHAR_BIT );
            auto constexpr nStepsBetweenOverflowChecks = boxSizeCudaType / 2 - 1;
            if ( iStep != 0 && iStep % nStepsBetweenOverflowChecks == 0 ){
                findAndRemoveOverflows( false );
                CUDA_ERROR( hipMemcpyAsync( mPolymerSystemSortedOld->gpu,
                    mPolymerSystemSorted->gpu, mPolymerSystemSortedOld->nBytes,
                    hipMemcpyDeviceToDevice, mStream ) );
            }
        }
        /* one Monte-Carlo step:
         *  - tries to move on average all particles one time
         *  - each particle could be touched, not just one group */
        for ( uint32_t iSubStep = 0; iSubStep < nSpecies; ++iSubStep ) 
		{
            // uint32_t iSubStep = 0;
            auto const iStepTotal = iStep * nSpecies + iSubStep;
            auto  iOffsetLatticeTmp = ( iStepTotal % mnLatticeTmpBuffers )
            * ( mBoxX * mBoxY * mBoxZ * sizeof( mLatticeTmp->gpu[0] ));
            if (met.getPacking().getBitPackingOn()) 
                iOffsetLatticeTmp /= CHAR_BIT;
            auto texLatticeTmp = mvtLatticeTmp[ iStepTotal % mnLatticeTmpBuffers ];
            if (met.getPacking().getNBufferedTmpLatticeOn()) {
                    iOffsetLatticeTmp = 0u;
                    texLatticeTmp = mLatticeTmp->texture;
            }
            /* randomly choose which monomer group to advance */
            auto const iSpecies = randomNumbers.r250_rand32() % nSpecies;
            auto const seed     = randomNumbers.r250_rand32();
            auto const nThreads = chooseThreads.getBestThread(iSpecies);
            auto const nBlocks  = ceilDiv( mnElementsInGroup[ iSpecies ], nThreads );
            // auto const useCudaMemset = chooseThreads.useCudaMemset(iSpecies);
            chooseThreads.addRecord(iSpecies, mStream);
            nSpeciesChosen[ iSpecies ] += 1;
            // if (!diagMovesOn)
            this-> template launch_CheckSpecies<6>(nBlocks, nThreads, iSpecies, iOffsetLatticeTmp, seed);
            // 	this-> template launch_CheckSpecies<18>(nBlocks, nThreads, iSpecies, iOffsetLatticeTmp, seed);
            launch_CheckSpeciesInteraction(nBlocks, nThreads, iSpecies,seed );
            // launch_resetInteractionLattice(nBlocks,nThreads,iSpecies);
            launch_PerformSpeciesAndApply(nBlocks, nThreads, iSpecies, texLatticeTmp );
            launch_ApplyInteraction(nBlocks, nThreads, iSpecies);
            // checkInteractionLatticeOccupation();
            // launch_initializeInteractionLattice(nBlocks,nThreads,iSpecies);
			
            if(met.getPacking().getNBufferedTmpLatticeOn()){
                /* we only need to delete when buffers will wrap around and
                    * on the last loop, so that on next runSimulationOnGPU
                    * call mLatticeTmp is clean */
                if ( ( iStepTotal % mnLatticeTmpBuffers == 0 ) ||
                    ( iStep == nMonteCarloSteps-1 && iSubStep == nSpecies-1 ) ){
                    hipMemsetAsync( (void*) mLatticeTmp->gpu, 0, mLatticeTmp->nBytes, mStream );
                }
            }else
                mLatticeTmp->memsetAsync(0);
            chooseThreads.analyze(iSpecies,mStream);
		} // iSubstep
    } // iStep
    CUDA_ERROR( hipStreamSynchronize( mStream ) );
    std::clock_t const t1 = std::clock();
    double const dt = float(t1-t0) / CLOCKS_PER_SEC;
    mLog( "Info" )
        << "run time (GPU): " << nMonteCarloSteps << "\n"
        << "mcs = " << nMonteCarloSteps  << "  speed [performed monomer try and move/s] = MCS*N/t: "
        << nMonteCarloSteps * ( mnAllMonomers / dt )  << "     runtime[s]:" << dt << "\n";
	checkSystem(); // no-op if "Check"-level deactivated
    checkInteractionLatticeOccupation();
    CUDA_ERROR( hipStreamSynchronize( mStream ) );
    BaseClass::doCopyBack();
    // if (mLog.isActive( "Check" ) )
    
}
////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
template class UpdaterGPU_Interaction< uint8_t  >;
template class UpdaterGPU_Interaction< uint16_t >;
template class UpdaterGPU_Interaction< uint32_t >;
template class UpdaterGPU_Interaction<  int16_t >;
template class UpdaterGPU_Interaction<  int32_t >;
////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////